#include "hip/hip_runtime.h"
﻿//Jeremy Udarbe
//CS 475
//Project 5 - CUDA Monte Carlo
#define _USE_MATH_DEFINES
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE 128 // number of threads in each block
#endif
#ifndef NUMTRIALS // WARNING: DON’T CALL THIS “ARRAYSIZE” !
#define NUMTRIALS ( 8*1024*1024 ) // size of the array
#endif
float hA[DATASET_SIZE];
float hB[DATASET_SIZE];
float hC[DATASET_SIZE];

void    CudaCheckError();
void	TimeOfDaySeed();
float Ranf(float low, float high);
int Ranf(int ilow, int ihigh);

//global variables
const float GRAVITY = -9.8;	// acceleraion due to gravity in meters / sec^2

// degrees-to-radians -- callable from the device:
__device__ float Radians(float d) {
    return (M_PI / 180.f) * d;
}

// the kernel:
__global__ void MonteCarlo(float* dvs, float* dths, float* dgs, float* dhs, float* dds, int* dhits) {
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // randomize everything:
    float v = dvs[gid];
    float thr = Radians(dths[gid]);
    float vx = v * cos(thr);
    float vy = v * sin(thr);
    float  g = dgs[gid];
    float  h = dhs[gid];
    float  d = dds[gid];

    int numHits = 0;

    // see if the ball doesn't even reach the cliff:
    float t = -vy / (0.5 * GRAVITY);
    float x = vx * t;
    if (x > g) {
        ...
            numHits = 1;
    }

    dhits[gid] = numHits;
}


// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT

int main(int argc, char* argv[]) {
    TimeOfDaySeed();

    int dev = findCudaDevice(argc, (const char**)argv);

    // better to define these here so that the rand() calls don't get into the thread timing:
    float* hvs = new float[NUMTRIALS];
    float* hths = new float[NUMTRIALS];
    float* hgs = new float[NUMTRIALS];
    float* hhs = new float[NUMTRIALS];
    float* hds = new float[NUMTRIALS];
    int* hhits = new int[NUMTRIALS];

    // fill the random-value arrays:

    ? ? ? ? ?


        // allocate device memory:
        float* dvs, * dths, * dgs, * dhs, * dds;
    int* dhits;

    hipMalloc(&dvs, NUMTRIALS * sizeof(float));
    hipMalloc(&dths, NUMTRIALS * sizeof(float));
    hipMalloc(&dgs, NUMTRIALS * sizeof(float));
    hipMalloc(&dhs, NUMTRIALS * sizeof(float));
    hipMalloc(&dds, NUMTRIALS * sizeof(float));
    hipMalloc(&dhits, NUMTRIALS * sizeof(int));
    CudaCheckError();

    // copy host memory to the device:
    hipMemcpy(dvs, hvs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dths, hths, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dgs, hgs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dhs, hhs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dds, hds, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    CudaCheckError();

    // setup the execution parameters:
    dim3 grid(NUMBLOCKS, 1, 1);
    dim3 threads(BLOCKSIZE, 1, 1);

    // allocate cuda events that we'll use for timing:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    CudaCheckError();

    // let the gpu go quiet:
    hipDeviceSynchronize();

    // record the start event:
    hipEventRecord(start, NULL);
    CudaCheckError();

    // execute the kernel:
    MonteCarlo << < grid, threads >> > (IN dvs, IN dths, IN dgs, IN dhs, IN dds, OUT dhits);

    // record the stop event:
    hipEventRecord(stop, NULL);
    CudaCheckError();

    // wait for the stop event to complete:
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    CudaCheckError();

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    CudaCheckError();

    // compute and print the performance

    ? ? ? ? ?

        // copy result from the device to the host:
        hipMemcpy(hhits, dhits, NUMTRIALS * sizeof(int), hipMemcpyDeviceToHost);
    CudaCheckError();

    // add up the hhits[ ] array: :

    ? ? ? ? ?

        // compute and print the probability:

        ? ? ? ? ?

        // clean up host memory:
        delete[] hvs;
    delete[] hths;
    delete[] hgs;
    delete[] hhs;
    delete[] hds;
    delete[] hhits;

    // clean up device memory:
    hipFree(dvs);
    hipFree(dths);
    hipFree(dgs);
    hipFree(dhs);
    hipFree(dds);
    hipFree(dhits);
    CudaCheckError();

    return 0;
}

void CudaCheckError() {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        fprintf(stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
    }
}

void TimeOfDaySeed() {
    struct tm y2k = { 0 };
    y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
    y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

    time_t  timer;
    time(&timer);
    double seconds = difftime(timer, mktime(&y2k));
    unsigned int seed = (unsigned int)(1000. * seconds);    // milliseconds
    srand(seed);
}

float Ranf(float low, float high) {
    float r = (float)rand();               // 0 - RAND_MAX
    float t = r / (float)RAND_MAX;       // 0. - 1.

    return   low + t * (high - low);
}

int Ranf(int ilow, int ihigh) {
    float low = (float)ilow;
    float high = ceil((float)ihigh);

    return (int)Ranf(low, high);
}
